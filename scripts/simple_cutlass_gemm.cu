#include <iostream>
#include <vector>
#include "CutlassGemm.cuh"
#include <hip/hip_runtime.h>
#include <cutlass/numeric_types.h>
#include <iterator>
#include <algorithm>

int main() {
    float x = 3.0f, y = 4.0f;
    float z = test_dot(x, y);
    std::cout << "x * y = " << z << std::endl;


    dim3 gridDim = (1);
    dim3 blockDim = (32);

    int N = 32;

    using half = cutlass::half_t;

    std::vector<half> A(N, half(1.0f));
    std::vector<half> B(N, half(2.0f));
    std::vector<half> C(N, half(0.0f));

    half *A_d, *B_d, *C_d;
    hipMalloc(&A_d, N * sizeof(half));
    hipMalloc(&B_d, N * sizeof(half));
    hipMalloc(&C_d, N * sizeof(half));

    hipMemcpy(A_d, A.data(), N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B.data(), N * sizeof(half), hipMemcpyHostToDevice);

    device_add_half<<<gridDim, blockDim>>>(A_d, B_d, C_d, N);
    hipDeviceSynchronize();

    hipMemcpy(C.data(), C_d, N * sizeof(half), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << static_cast<float>(C[i]) << " ";
    }
    std::cout << std::endl;

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    return 0;
}